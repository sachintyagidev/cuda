#include "hip/hip_runtime.h"
#include "um.h"
#include <stdio.h>
#include <string>
#include <vector>
#include <iostream>
#include <list>

#define N 1024

__global__ void kernel(int *a) {
	a[threadIdx.x]= threadIdx.x + 1;
}

int main() {
	UM umObj;
	int i;
	int *a = (int*)malloc(N*sizeof(int));

	/*Move this on Struct based on reflection (reflection ts)*/
	umObj.alocateMemObject<int, float>(N, N);

	int *d_a = (int *)umObj.getRefrence(0);
	float *d_a_1 = (float *)umObj.getRefrence(1);

	umObj.prerfetchOnDevice(0, N);

	umObj.prerfetchOnHost(1, N);

	umObj.setReadOnlyPrefer(1, N);

	umObj.setPreferAsHost(1, N);
	
	umObj.setPreferAsDevice(0, N);

	kernel<<<1, N>>>(d_a);

	hipMemcpy(a, d_a, N*sizeof(int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	for(i=0;i<N;i++) {
			printf("%d ",a[i]);
	}

	return 0;
}